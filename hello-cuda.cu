
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__
void helloKernel(int N)
{
  printf("hello from GPU\n");
  int i = threadIdx.x + blockIdx.x*blockDim.x;
  if (i<N) {
    printf("thread %d of block %d (dim: %d): iter %d of %d\n", threadIdx.x, blockIdx.x, blockDim.x, i, N);
  }
}

int main(void) {
  int N=10,numGPUs;

  /* check have GPU else quit */
  hipGetDeviceCount(&numGPUs);
  printf("Number of  GPU = %d\n", numGPUs);
  if (numGPUs >= 1 ) {
  printf("hello on CPU\n");

  /* call GPU kernel  */
  helloKernel<<<4,3>>> (N);

  /* Synchronize */
  hipDeviceSynchronize();

  printf("hello again from CPU\n");
  
  }
  else {
   printf("no GPU present\n");
   }
}


