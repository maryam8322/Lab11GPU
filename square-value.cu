
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

int N =10;

__global__ void squareKernel(int N) {
  int i = threadIdx.x + blockIdx.x*blockDim.x;
  if (i < N) {
    int square = i * i;
    printf("GPU thread %d of block %d (dim: %d): value %d, square %d\n", threadIdx.x, blockIdx.x, blockDim.x, i, square);
  }
}


int main(void) {
  int numGPUs;
  printf("hello from CPU\n");
  
  /* check have GPU else quit */
  hipGetDeviceCount(&numGPUs);
  
  if (numGPUs >= 1 ) {
  /* Synchronize */
  hipDeviceSynchronize();
  /* call GPU kernel  */
  squareKernel<<<4,3>>> (N);
  /* Synchronize */
  hipDeviceSynchronize();
  }
  
  else {
   printf("no GPU present\n");
   }
   printf("hello again from CPU\n");
}


